//
//Jake Shankman 4/22/2013
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//
#define N 7

__global__ void jacobi(int* bcuda, float* xcuda, float* x1)
{
  int rank = threadIdx.x; //flat model
  //
    if(rank - 1 < 0)
      {
	x1[rank] = (1.0f/3)*(bcuda[rank] - (-1*xcuda[rank + 1]));
      }
    else if(rank + 1 > N)
      {
	x1[rank] = (1.0f/3)*(bcuda[rank] - (-1*xcuda[rank - 1]));
      }
    else
      {
      x1[rank] = (1.0f/3)*(bcuda[rank] - (-1*xcuda[rank-1] + -1*xcuda[rank + 1]));
      }
}
int allOne(float* matrix)
{
  int i;
  for(i = 0; i < N; ++i)
  {
    if(matrix[i] != 1.0f) // matrix[i] is the same as *(matrix + i)
      return 0;
  }
  return 1;
}
//
int main(void)
{
  int b[N];
  int *bcuda;
  hipMalloc((void**)&bcuda, sizeof(int)*N);
  //float x0[N], x1[N];
  float *x0 = (float*) malloc(sizeof(float) * N);
  float *x1;
  float *xcuda;
  float *xtmp;
  int i;
  printf("x0: [");
  for (i = 0; i < N; ++i)
  {
    if(i == 0 || i == N-1)
    {
      b[i] = 2;
    }
    else
    {
      b[i] = 1;
    }
    x0[i] = rand()/(float)RAND_MAX;
    printf("%f ", x0[i]);
  }
  printf("]\n");
  hipMalloc((void**)&xcuda, sizeof(float)*N);
  hipMemcpy(bcuda, b, sizeof(int)*N, hipMemcpyHostToDevice);
  //xi+1 = Dinverse(b - (R)xi)
  int count = 1;
  while(allOne(x0) != 1)
  {
    hipMemcpy(xcuda, x0, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMalloc((void**)&x1, sizeof(float)*N);
    printf("x%d: [", count);
    //
    dim3 dimGrid(1), dimBlock(N);
    //
    //do following for loop in parallel
    jacobi<<<dimGrid,dimBlock>>>(bcuda,xcuda, x1);
    xtmp = xcuda;
    xcuda = x1;
    hipFree(xtmp);
    hipMemcpy(x0, xcuda, sizeof(float)*N, hipMemcpyDeviceToHost);    
    for(i = 0; i < N; ++i)
    {
      printf("%f ", x0[i]);
    }   
    printf("]\n");
    ++count;
  }
  return 0;
}